#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void ddpm_update(float* x, float *eps, float *out, float alpha, float beta, float alpha_bar, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        float inv_sqrt_alpha = 1.0f / sqrtf(alpha);
        float scale_eps = beta / sqrtf(1.0f - alpha_bar);
        out[idx] = inv_sqrt_alpha * (x[idx] - scale_eps * eps[idx]);
    }
}

int main(){
    int n = 1024 * 1024 * 3;
    float alpha = 0.9f, beta = 0.1f, alpha_bar = 0.5f;

    float *x = (float*)malloc(n * sizeof(float));
    float *eps = (float*)malloc(n * sizeof(float));
    float *out = (float*)malloc(n * sizeof(float));

    for(int i = 0; i < n; i++){
        x[i] = ((float)rand() / RAND_MAX) * 2 - 1;
        eps[i] = ((float)rand() / RAND_MAX) * 2 - 1;
    }

    float *dx, *deps, *dout;
    hipMalloc(&dx, n * sizeof(float));
    hipMalloc(&deps, n * sizeof(float));
    hipMalloc(&dout, n * sizeof(float));

    hipMemcpy(dx, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deps, eps, n * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 1024;
    int grid_size = (n + block_size - 1) / block_size;

    hipDeviceSynchronize();
    float ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for(int i = 0; i < 10000; i++){
        ddpm_update<<<grid_size, block_size>>>(dx, deps, out, alpha, beta, alpha_bar, n);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    printf("Cuda Kernel Time: %f ms\n", ms / 1000.0);

    hipMemcpy(out, dout, n * sizeof(float), hipMemcpyDeviceToHost);


    hipFree(dx);
    hipFree(deps);
    hipFree(dout);

    
    return 0;
}