#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

void init_data(float* data, int x, int y) {
    for (int i = 0; i < x; i++) {
        for (int j = 0; j < y; j++) {
            data[i * y + j] = i + j;  // row-major: data[row * cols + col]
        }
    }
}

void print_mat(float* data, int x, int y) {
    for (int i = 0; i < x; i++) {
        for (int j = 0; j < y; j++) {
            printf("%f ", data[i * y + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    hipblasHandle_t handle;
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS initialization failed!\n");
        return -1;
    }

    int M = 2, N = 3, K = 4;  
    float *a, *b, *c;
    a = (float*)malloc(M * K * sizeof(float));  
    b = (float*)malloc(K * N * sizeof(float));  
    c = (float*)malloc(M * N * sizeof(float));  

    init_data(a, M, K);
    init_data(b, K, N);

    float *da, *db, *dc;
    hipMalloc(&da, M * K * sizeof(float));
    hipMalloc(&db, K * N * sizeof(float));
    hipMalloc(&dc, M * N * sizeof(float));

    hipMemcpy(da, a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, K * N * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f, beta = 0.0f;

    // C = alpha * op(A) * op(B) + beta * C
    hipblasStatus_t stat = hipblasSgemm(
        handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T,  
        M, N, K,
        &alpha,
        da, K,  
        db, N,  
        &beta,
        dc, M   
    );

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS SGEMM failed!\n");
        return -1;
    }

    hipMemcpy(c, dc, M * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Mat A:\n");
    print_mat(a, M, K);

    printf("Mat B:\n");
    print_mat(b, K, N);

    printf("Mat C:\n");
    print_mat(c, M, N);

    hipFree(da); hipFree(db); hipFree(dc);
    free(a); free(b); free(c);
    hipblasDestroy(handle);

    return 0;
}
