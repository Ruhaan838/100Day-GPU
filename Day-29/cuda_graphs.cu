#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <iostream>

using namespace std;


void cuda_check(hipError_t err, string msg){
    if (err != hipSuccess){
        cout << "Cuda Error at "  << __FILE__ << __LINE__ << hipGetErrorString(err) << " For " << msg << "\n";
        exit(EXIT_FAILURE);
    }
}

const int N = 100000;
const int NUM_ITERATIONS = 10000;
const int BLOCK_SIZE = 256;

__global__ void matrixAdd(const float* a, const float* b, float* c, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

__global__ void matrixScale(float* a, float scale, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        a[idx] = a[idx ] * scale;
}

__global__ void matrixSquare(float* a, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        a[idx] = a[idx] * a[idx];
}

__global__ void matrixOffset(float* a, float offset, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        a[idx] = a[idx] + offset; 
}

void Result_varification(float* a, float* b, float* c, float* verify, int n){
    for (int i = 0; i < n; i++){
        float temp = a[i] + b[i];
        temp = temp * 2.0f;
        temp = temp * temp;
        verify[i] = temp + 1.0f;
    }

    bool match = true;
    for (int i = 0; i < n; i++){
        if (abs(verify[i] - c[i]) > 1e-5){
            match = false;
            printf("Mismatch at idx %d: Expected %f, got %f\n", i, verify[i], c[i]);
            break;
        }
    }

    if (match)
        printf("All values match not a single problem\n");
}

int main(){
    float *a, *b, *c, *verify;
    float *da, *db, *dc;
    size_t size = N * sizeof(float);

    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);
    verify = (float*)malloc(size);

    for (int i = 0; i < N; i++){
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
    }

    cuda_check(hipMalloc(&da, size), "allocate da");
    cuda_check(hipMalloc(&db, size), "allocate db");
    cuda_check(hipMalloc(&dc, size), "allocate dc");

    cuda_check(hipMemcpy(da, a, size, hipMemcpyHostToDevice), "da -> a");
    cuda_check(hipMemcpy(db, b, size, hipMemcpyHostToDevice), "db -> b");

    hipStream_t stream;
    hipEvent_t start, stop;
    cuda_check(hipStreamCreate(&stream), "Stream create");
    cuda_check(hipEventCreate(&start), "Stream start");
    cuda_check(hipEventCreate(&stop), "Stream stop");


    dim3 grid_size((N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    for (int i = 0; i < 10; i++){
        matrixAdd<<<grid_size, BLOCK_SIZE, 0, stream>>>(da, db, dc, N);
        matrixScale<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, 2.0f, N);
        matrixSquare<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, N);
        matrixOffset<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, 1.0f, N);
    }
    hipStreamSynchronize(stream);

    hipEventRecord(start, stream);
    for (int i = 0; i < NUM_ITERATIONS; i++){
        matrixAdd<<<grid_size, BLOCK_SIZE, 0, stream>>>(da, db, dc, N);
        matrixScale<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, 2.0f, N);
        matrixSquare<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, N);
        matrixOffset<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, 1.0f, N);
    }

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);

    float mil_s = 0;
    hipEventElapsedTime(&mil_s, start, stop);
    printf("Without CUDA Graphs: %.3f ms\n", mil_s);

    cuda_check(hipMemcpy(c, dc, size, hipMemcpyDeviceToHost), "c -> dc");

    hipGraph_t graph;
    hipGraphExec_t graphExec;

    cuda_check(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal), "stream capture");
    matrixAdd<<<grid_size, BLOCK_SIZE, 0, stream>>>(da, db, dc, N);
    matrixScale<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, 2.0f, N);
    matrixSquare<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, N);
    matrixOffset<<<grid_size, BLOCK_SIZE, 0, stream>>>(dc, 1.0f, N);
    cuda_check(hipStreamEndCapture(stream, &graph), "Stream End capture");
    cuda_check(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0), "graph instatiante");

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&mil_s, start, stop);
    printf("With CUDA Graphs: %.3f ms\n", mil_s);

    cuda_check(hipMemcpy(c, dc, size, hipMemcpyDeviceToHost), "c - > dc");
    Result_varification(a, b, c, verify, N);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    free(verify);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);

    return 0;
}