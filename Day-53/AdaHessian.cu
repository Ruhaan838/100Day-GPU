#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void adaHessianKernel(
    float* theta,
    const float* grad,
    const float* gradPerturbed,
    float* m_moment,
    float* v_moment,
    const float lr,
    const float beta1, 
    const float beta2,
    const float eps,
    const float delta,
    int N
){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N) return;

    float h_diag = (gradPerturbed[idx] - grad[idx]) / delta;
    m_moment[idx] = beta1 * m_moment[idx] + (1.0f - beta1) * grad[idx];
    v_moment[idx] = beta2 * v_moment[idx] + (1.0f - beta2) * powf(h_diag, 2);

    theta[idx] -= lr * m_moment[idx] / (sqrtf(v_moment[idx]) + eps);
}

int main(){
    const int N = 10;
    size_t size = N * sizeof(float);
    const float lr = 0.01f;
    const float beta1 = 0.9f;
    const float beta2 = 0.999f;
    const float eps = 1e-7f;
    const float delta = 1e-4f;

    float theta[N], grad[N], gradPerturbed[N], m_moment[N], v_moment[N];

    for(int i = 0; i < N; i++){
        theta[i] = 1.0f;
        grad[i] = 0.1f;
        gradPerturbed[i] = 0.1f + 0.001f * i;
        m_moment[i] = 0.0f;
        v_moment[i] = 0.0f;
    }

    float *d_theta, *d_grad, *d_gradPerturbed, *d_m_moment, *d_v_moment;
    hipMalloc((void**)&d_theta, size);
    hipMalloc((void**)&d_grad, size);
    hipMalloc((void**)&d_gradPerturbed, size);
    hipMalloc((void**)&d_m_moment, size);
    hipMalloc((void**)&d_v_moment, size);

    hipMemcpy(d_theta, theta, size, hipMemcpyHostToDevice);
    hipMemcpy(d_grad, grad, size, hipMemcpyHostToDevice);
    hipMemcpy(d_gradPerturbed, gradPerturbed, size, hipMemcpyHostToDevice);
    hipMemcpy(d_m_moment, m_moment, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v_moment, v_moment, size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    adaHessianKernel<<<grid_size, block_size>>>(
        d_theta, d_grad, d_gradPerturbed, d_m_moment, d_v_moment,
        lr, beta1, beta2, eps, delta, N
    );

    hipDeviceSynchronize();

    hipMemcpy(theta, d_theta, size, hipMemcpyDeviceToHost);
    hipMemcpy(m_moment, d_m_moment, size, hipMemcpyDeviceToHost);
    hipMemcpy(v_moment, d_v_moment, size, hipMemcpyDeviceToHost);

    printf("Updated theta Values:\n");
    for(int i = 0; i < N; i++){
        printf("%f ", theta[i]);
    }
    printf("\n");

    hipFree(d_theta);
    hipFree(d_grad);
    hipFree(d_gradPerturbed);
    hipFree(d_m_moment);
    hipFree(d_v_moment);

    return 0;
}