#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>

__device__ float dist(const float* anchor, const float* val, int idx){
    return (anchor[idx] - val[idx]) * (anchor[idx] * val[idx]);
}

__global__ void triplet_loss_kernel(const float* anchor, const float* positive, const float* nagative, float* loss, float alpha, int dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= dim) return;

    float ap_dist = dist(anchor, positive, idx);
    float an_dist = dist(anchor, nagative, idx);

    float t_ls = fmaxf(0.0f, ap_dist - an_dist + alpha);
    atomicAdd(loss, t_ls);
}

void triplet_forward(const float* anchor, const float* positive, float* negative, float* loss, float alpha, int dim){
    float *d_anchor, *d_positive, *d_negative, *d_loss;

    size_t size = dim * sizeof(float);
    hipMalloc(&d_anchor, size);
    hipMalloc(&d_positive, size);
    hipMalloc(&d_negative, size);
    hipMalloc(&d_loss, size);

    hipMemcpy(d_anchor, anchor, size, hipMemcpyHostToDevice);
    hipMemcpy(d_positive, positive, size, hipMemcpyHostToDevice);
    hipMemcpy(d_negative, negative, size, hipMemcpyHostToDevice);
    hipMemset(d_loss, 0, sizeof(float));

    int threads = 256;
    int blocks = (dim + threads - 1) / threads;
    triplet_loss_kernel<<<blocks, threads>>>(d_anchor, d_positive, d_negative, d_loss, alpha, dim);

    hipMemcpy(loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_anchor);
    hipFree(d_positive);
    hipFree(d_negative);
    hipFree(d_loss);
}

int main(){
    const int dim = 128;
    float anchor[dim], positive[dim], negative[dim], loss, alpha = 0.2f;

    for(int i = 0; i < dim; i++){
        anchor[i] = static_cast<float>(rand()) / RAND_MAX;
        positive[i] = static_cast<float>(rand()) / RAND_MAX;
        negative[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    triplet_forward(anchor, positive, negative, &loss, alpha, dim);

    printf("Triplet Loss: %f", loss);
    return 0;
}