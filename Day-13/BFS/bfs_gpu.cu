#include "bfs.h"
#include "bfs_kernel.cu"

void bfs_gpu(int source, int num_vertex, int num_edgs, int* h_edgs, int* h_dest, int* h_labels){
    int *d_edgs, *d_dest, *d_labels, *d_done;

    CUDA_ERROR(hipMalloc((void**)&d_edgs, (num_vertex + 1) * sizeof(int)));
    CUDA_ERROR(hipMalloc((void**)&d_dest, num_edgs * sizeof(int)));
    CUDA_ERROR(hipMalloc((void**)&d_labels, num_vertex * sizeof(int)));
    CUDA_ERROR(hipMalloc((void**)&d_done, sizeof(int)));

    CUDA_ERROR(hipMemset(d_labels, -1, num_vertex * sizeof(int)));

    CUDA_ERROR(hipMemcpy(d_edgs, h_edgs, (num_vertex + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_ERROR(hipMemcpy(d_dest, h_dest, num_edgs * sizeof(int), hipMemcpyHostToDevice));

    int init_level = 0;
    CUDA_ERROR(hipMemcpy(d_labels + source, &init_level, sizeof(int), hipMemcpyHostToDevice));

    int level = 0;
    int h_done;
    int thread_block = THREADS_PER_BLOCK;
    int block_per_grid = (num_vertex + thread_block - 1) / thread_block;

    do {
        h_done = 1;
        CUDA_ERROR(hipMemcpy(d_done, &h_done, sizeof(int), hipMemcpyHostToDevice));

        bfs_kernel<<<block_per_grid, thread_block>>>(level, num_vertex, d_edgs, d_edgs, d_labels, d_done);
        CUDA_ERROR(hipDeviceSynchronize());

        CUDA_ERROR(hipMemcpy(&h_done, d_done, sizeof(int)), hipMemcpyDeviceToHost);
        level++;
    } while (!h_done && level < num_vertex);

    CUDA_ERROR(hipMemcpy(h_labels, d_labels, num_vertex * sizeof(int)), hipMemcpyDeviceToHost);

    CUDA_ERROR(hipFree(d_edgs));
    CUDA_ERROR(hipFree(d_dest));
    CUDA_ERROR(hipFree(d_labels));
    CUDA_ERROR(hipFree(d_done));
    

}