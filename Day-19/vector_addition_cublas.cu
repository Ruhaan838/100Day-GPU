
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

using namespace std;

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    for(int i = 0; i < N; i++){
        A[i] = i; B[i] = i;
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    size_t vec_bit  = N * sizeof(float);
    //alocate cuda memeory
    float *da, *db;
    hipMalloc(&da, vec_bit);
    hipMalloc(&db, vec_bit);

    hipMemcpy(da, A, vec_bit, hipMemcpyHostToDevice);
    hipMemcpy(db, B, vec_bit, hipMemcpyHostToDevice);

    const float alpha = 1.0f;

    //perfroming addition
    // 
    hipblasSaxpy(handle, N, &alpha, da, 2, db, 2);

    hipMemcpy(C, db, vec_bit, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++)
        cout << C[i] << " ";
    
    cout << '\n';

    hipFree(da);
    hipFree(db);
    hipblasDestroy(handle);

    return 0;
}