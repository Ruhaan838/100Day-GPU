
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_vector(const float* a, float* b, float* c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] + b[i];
    }
}

int main(){
    const int N = 10;
    float a[N], b[N], c[N];

    float *da, *db, *dc;
    hipMalloc(&da, N*sizeof(float));
    hipMalloc(&db, N*sizeof(float));
    hipMalloc(&dc, N*sizeof(float));

    hipMemcpy(da, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, N*sizeof(float), hipMemcpyHostToDevice);

    int blocksize = 256;
    int gridsize = ceil(N/blocksize);

    add_vector<<<gridsize, blocksize>>>(da, db, dc, N);

    hipMemcpy(c, dc, N*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}