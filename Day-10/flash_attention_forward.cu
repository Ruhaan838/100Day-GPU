#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
#include <cmath>
#include <hiprand/hiprand.h>
#include <fstream>

using namespace std;

__global__ void FlashAttentionForward(
    const float* query_ptr,
    const float* key_ptr,
    const float* value_ptr,
    float *output,

    const int seq_len,
    const int embd_dim,

    const int col_block_size,
    const int row_block_size,

    const int total_row_size,
    const int total_col_size,

    const float attention_scale,

    float* sum_mat,
    float* max_mat
) {
    int thread_x = threadIdx.x;
    int block_idx_x = blockIdx.x;
    int block_idx_y = blockIdx.y;


    // the gridDim.y is num_heads 
    // we perfrom the attention for the (b, s, n_heads, d_k) 
    // where: d_k is embd_dim // n_heads
    int qkv_offset = (block_idx_x * gridDim.y * seq_len * embd_dim) + (block_idx_y * seq_len * embd_dim);
    //offset for (sum/max) mat
    int lm_offset = (block_idx_x * gridDim.y * seq_len) + (block_idx_y * seq_len); 

    //define the dynamic shared RAM for the QKV and attention_score
    extern __shared__ float shared_memory[];
    int tile_size = col_block_size * embd_dim;
    float* query_shared = shared_memory;
    float* key_shared = &shared_memory[tile_size];
    float* value_shared = &shared_memory[tile_size * 2];
    float* attention_shared = &shared_memory[tile_size * 3];

    float eps = 1e-10;

    //load the Key Value to shared memeory
    for (int col = 0; col < total_col_size; col++){
        // loading the key and value
        for (int embd = 0; embd < embd_dim; embd++){
            int temp_shared_offset = (thread_x * embd_dim) + embd;
            int temp_mat_offset = (qkv_offset + (tile_size * col) + temp_shared_offset);
            key_shared[(temp_shared_offset)] = key_ptr[temp_mat_offset];
            value_shared[(temp_shared_offset)] = value_ptr[temp_mat_offset];
        }

        __syncthreads();

        for (int row = 0; row < total_row_size; row++){
            // loading the query data rowwise 
            for (int embd = 0; embd < embd_dim; embd++)
                query_shared[(thread_x * embd_dim) + embd] = query_ptr[qkv_offset + (tile_size * row) + (thread_x * embd_dim) + embd];

            // this reduce offset access the data from the max and sum mat 
            int reduce_offset = lm_offset + (row_block_size * row) + thread_x;
            float row_max = max_mat[reduce_offset];
            float row_sum = sum_mat[reduce_offset];

            float row_max_new = -1e20; // init the max row with big number
            for (int col_inner = 0; col_inner < col_block_size; col_inner++){
                float sum = 0; // use the cached sum method
                // perfroming the Q @ K.T
                for (int embd = 0; embd < embd_dim; embd++)
                    sum += query_shared[(thread_x * embd_dim) + embd] * key_shared[(col_inner * embd_dim) + embd];
                // pefroming the Q @ K.T / sqrt(d_k)
                sum *= attention_scale;
                // update the attention_score
                attention_shared[(col_block_size * thread_x) + col_inner] = sum;

                if (sum > row_max_new)
                    row_max_new = sum;

            }

            float row_sum_new = 0; // row_sum_new = sum(exp(attention_score - row_max))
            for (int col_inner = 0; col_inner < col_block_size; col_inner++){
                int temp_col_offset = (col_block_size * thread_x) + col_inner;
                // exp(attention_score - row_max)
                attention_shared[temp_col_offset] = __expf(attention_shared[temp_col_offset] - row_max);
                // sum the exp of the attention_score
                row_sum_new += attention_shared[temp_col_offset];
            }
            
            float row_max_f = max(row_max, row_max_new);
            float row_sum_f = (__expf(row_max - row_max_f) * row_sum) + (__expf(row_max_new - row_max_f) * row_sum_new);

            //write the output and sum_mat, and max_mat
            for (int embd = 0; embd < embd_dim; embd++){
                float weight = 0;
                for (int col_inner = 0; col_inner < col_block_size; col_inner++)
                    weight += attention_shared[(col_block_size * thread_x) + col_inner] * value_shared[(col_inner * embd_dim) + embd] + eps;
                
                // output_ij = (1 / new_sum) * (old_sum * e ^ (old_max - new_max) + e ^ attention_score - new_max) * weight
                // this output formula is actually numerically stable that's why it's takes more formula
                int output_offset = qkv_offset + (tile_size * row) + (thread_x * embd_dim) + embd;
                output[output_offset] = (1 / (eps + row_sum_f)) * ((row_sum * __expf(row_max - row_max_f) * output[output_offset])) + __expf(row_max_new - row_max_f + eps) * weight;
            }
            int new_reduce_offset = lm_offset + (row_block_size * row) + thread_x;
            max_mat[new_reduce_offset] = row_max_f;
            sum_mat[new_reduce_offset] = row_sum_f;
        }
        __syncthreads();

    }
}

template <typename T>
T* allocate_init_cuda_memory(size_t size, bool init_with_zero = false, bool init_with_neg_inf = false){
    T* data_ptr;
    hipMalloc(&data_ptr, size);

    if (init_with_zero)
        hipMemset(data_ptr, 0, size);
    else if (init_with_neg_inf){
        float neg_inf = -INFINITY;
        hipMemset(data_ptr, *reinterpret_cast<int*>(&neg_inf), size);
    } else {
        hiprandGenerator_t generator;
        hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT); 
        hiprandSetGeneratorOffset(generator, time(0)); 
        hiprandGenerateUniform(generator, reinterpret_cast<float*>(data_ptr), size / sizeof(T));
        hiprandDestroyGenerator(generator);
    }

    return data_ptr;
}

template <typename T>
void write_mat_to_file(T* mat, const string& filename, int batch_size, int num_heads, int seq_len, int embd_dim){
    ofstream file(filename);
    if (!file){
        cerr << "Could not open the file!" << endl;
        return;
    }

    for (int b = 0; b < batch_size; ++b){
        for(int h = 0; h < num_heads; ++h){
            for(int s = 0; s < seq_len; ++s){
                for (int j = 0; j < embd_dim; ++j){
                    file << mat[(b * num_heads * seq_len * embd_dim) +
                         (h * seq_len * embd_dim) + 
                         (s * embd_dim) + j];
                    if (j < embd_dim - 1)
                        file << ", ";
                }
                file << endl;
            }
            file << endl;
        }
    }
    file.close();
}

template <typename T>
void print_matrix(T* mat, int batch_size, int num_heads, int seq_len, int embd_dim){
    int size = batch_size * num_heads * seq_len * embd_dim;
    T* host_mat = new T[size];
    hipMemcpy(host_mat, mat, size * sizeof(T), hipMemcpyDeviceToHost);

    cout << "Matrix:\n";
    for (int b = 0; b < batch_size; ++b){
        for(int h = 0; h < num_heads; ++h){
            for(int s = 0; s < seq_len; ++s){
                for (int j = 0; j < embd_dim; ++j){
                    cout << host_mat[(b * num_heads * seq_len * embd_dim) +
                         (h * seq_len * embd_dim) + 
                         (s * embd_dim) + j] << " ";
                }
                cout << endl;
            }
            cout << endl;
        }
    }
    delete[] host_mat;
}

int main(){
    const int batch_size = 1;
    const int num_heads = 1;
    const int seq_len = 64;
    const int embd_dim = 64;

    const int col_block_size = 32;
    const int row_block_size = 32;

    const int total_col_size = ceil((float)seq_len / col_block_size);
    const int total_row_size = ceil((float)seq_len / row_block_size);
    const float attention_scale = 1.0f / sqrt(embd_dim);

    size_t mat_size = batch_size * num_heads * seq_len * embd_dim * sizeof(float);
    size_t vector_size = batch_size * num_heads * seq_len * sizeof(float);

    float* query_mat = allocate_init_cuda_memory<float>(mat_size);
    float* key_mat = allocate_init_cuda_memory<float>(mat_size);
    float* value_mat = allocate_init_cuda_memory<float>(mat_size);

    float* output = allocate_init_cuda_memory<float>(mat_size, true);


    float* sum_mat = allocate_init_cuda_memory<float>(vector_size, true);
    float* max_mat = allocate_init_cuda_memory<float>(vector_size, false, true);

    const int shared_mem_size = (4 * col_block_size * embd_dim * sizeof(float)) + (col_block_size * row_block_size * sizeof(float));
    int max_shared_mem;
    hipDeviceGetAttribute(&max_shared_mem, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    
    dim3 grid_dim(batch_size, num_heads);
    dim3 block_dim(col_block_size);

    FlashAttentionForward<<<grid_dim, block_dim, shared_mem_size>>>(
        query_mat, key_mat, value_mat, output, seq_len, embd_dim, col_block_size, 
        row_block_size, total_col_size, total_row_size, attention_scale, 
        sum_mat, max_mat
    );

    hipDeviceSynchronize();
    int size = batch_size * num_heads * seq_len * embd_dim;
    float* query_host = new float[size];
    float* key_host = new float[size];
    float* value_host = new float[size];
    float* output_host = new float[size];

    hipMemcpy(query_host, query_mat, mat_size, hipMemcpyDeviceToHost);
    hipMemcpy(key_host, key_mat, mat_size, hipMemcpyDeviceToHost);
    hipMemcpy(value_host, value_mat, mat_size, hipMemcpyDeviceToHost);
    hipMemcpy(output_host, output, mat_size, hipMemcpyDeviceToHost);

    write_mat_to_file(query_host, "query_out.csv", batch_size, num_heads, seq_len, embd_dim);
    write_mat_to_file(key_host, "key_out.csv", batch_size, num_heads, seq_len, embd_dim);
    write_mat_to_file(value_host, "value_out.csv", batch_size, num_heads, seq_len, embd_dim);
    write_mat_to_file(output_host, "output.csv", batch_size, num_heads, seq_len, embd_dim);

    cout << "Query ";
    print_matrix(query_mat, batch_size, num_heads, seq_len, embd_dim);

    cout << "Key: ";
    print_matrix(query_mat, batch_size, num_heads, seq_len, embd_dim);

    cout << "Value: ";
    print_matrix(query_mat, batch_size, num_heads, seq_len, embd_dim);

    cout << "Output: ";
    print_matrix(query_mat, batch_size, num_heads, seq_len, embd_dim);

    hipFree(query_mat);
    hipFree(key_mat);
    hipFree(value_mat);
    hipFree(output);
    hipFree(sum_mat);
    hipFree(max_mat);

    return 0;

}