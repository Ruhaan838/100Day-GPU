#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void mseKernel(const float* preds, const float* tgt, size_t num_ele, float* sum){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= num_ele) return;
    float diff = preds[idx] - tgt[idx];
    float sq_diff = diff * diff;

    atomicAdd(sum, sq_diff);
}

int main(){
    const size_t num_ele = 10;
    float h_preds[num_ele] = {1, 1, 0, 1, 0, 1, 0, 1, 1, 0};
    float h_tgt[num_ele] = {1, 1, 1, 1, 0, 0, 0, 1, 0, 1};

    

    float *d_preds, *d_tgt, *d_sum;
    hipMalloc(&d_preds, num_ele * sizeof(float));
    hipMalloc(&d_tgt,   num_ele * sizeof(float));
    hipMalloc(&d_sum,   sizeof(float));

    hipMemcpy(d_preds, h_preds, num_ele * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tgt,   h_tgt,   num_ele * sizeof(float), hipMemcpyHostToDevice);

    float h_sum = 0.0f;
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (num_ele + threadsPerBlock - 1) / threadsPerBlock;
    mseKernel<<<blocks, threadsPerBlock>>>(d_preds, d_tgt, num_ele, d_sum);

    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    float mse = h_sum / static_cast<float>(num_ele);
    printf("MSE = %f\n", mse);

    hipFree(d_preds);
    hipFree(d_tgt);
    hipFree(d_sum);

    return 0;
}
